#include "hip/hip_runtime.h"
#include "common.h"

__global__ void dmem_kernel(float* ptr);

int main(int argc, char* argv[]) {
  Benchmark benchmark;

  // Parameters
  int maxThreadsPerBlock = benchmark.maxThreadsPerBlock();

  // Amount of work performed
  unsigned fetchPerBlock = 16;
  int maxItems = benchmark.totalGlobalMem() / sizeof(float) / 2;
  int numItems = roundToPowOf2(maxItems);
  double gbytes = (float)(numItems / fetchPerBlock) * sizeof(float) / 1e9;
  double gflops = 0;

  // Kernel dimensions
  dim3 grid(numItems / (fetchPerBlock * maxThreadsPerBlock));
  dim3 block(maxThreadsPerBlock);

  // Allocate memory
  benchmark.allocate(numItems * sizeof(float));

  // Run benchmark
  for (int i = 0; i < NR_BENCHMARKS; i++) {
    benchmark.run(reinterpret_cast<void*>(&dmem_kernel), grid, block, "dmem",
                  gflops, gbytes);
  }

  return EXIT_SUCCESS;
}
