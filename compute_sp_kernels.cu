#include "hip/hip_runtime.h"

#define FMA_1(x, y)  asm("fma.rn.f32 %0, %1, %2, %3;" : "=f"(x) : "f"(x), "f"(y), "f"(x)); \
                     asm("fma.rn.f32 %0, %1, %2, %3;" : "=f"(y) : "f"(y), "f"(x), "f"(y));
#define FMA_4(x, y)  FMA_1(x, y)  FMA_1(x, y)  FMA_1(x, y)  FMA_1(x,y)
#define FMA_16(x, y) FMA_4(x, y)  FMA_4(x, y)  FMA_4(x, y)  FMA_4(x, y)
#define FMA_64(x, y) FMA_16(x, y) FMA_16(x, y) FMA_16(x, y) FMA_16(x, y)

__global__ void compute_sp_v1(float *ptr)
{
    float x = threadIdx.x;
    float y = 0;

    for (int i = 0; i < 1024; i++) {
        FMA_64(x, y);   FMA_64(x, y);   FMA_64(x, y);   FMA_64(x, y);
        FMA_64(x, y);   FMA_64(x, y);   FMA_64(x, y);   FMA_64(x, y);
        FMA_64(x, y);   FMA_64(x, y);   FMA_64(x, y);   FMA_64(x, y);
        FMA_64(x, y);   FMA_64(x, y);   FMA_64(x, y);   FMA_64(x, y);
        FMA_64(x, y);   FMA_64(x, y);   FMA_64(x, y);   FMA_64(x, y);
        FMA_64(x, y);   FMA_64(x, y);   FMA_64(x, y);   FMA_64(x, y);
        FMA_64(x, y);   FMA_64(x, y);   FMA_64(x, y);   FMA_64(x, y);
        FMA_64(x, y);   FMA_64(x, y);   FMA_64(x, y);   FMA_64(x, y);

        FMA_64(x, y);   FMA_64(x, y);   FMA_64(x, y);   FMA_64(x, y);
        FMA_64(x, y);   FMA_64(x, y);   FMA_64(x, y);   FMA_64(x, y);
        FMA_64(x, y);   FMA_64(x, y);   FMA_64(x, y);   FMA_64(x, y);
        FMA_64(x, y);   FMA_64(x, y);   FMA_64(x, y);   FMA_64(x, y);
        FMA_64(x, y);   FMA_64(x, y);   FMA_64(x, y);   FMA_64(x, y);
        FMA_64(x, y);   FMA_64(x, y);   FMA_64(x, y);   FMA_64(x, y);
        FMA_64(x, y);   FMA_64(x, y);   FMA_64(x, y);   FMA_64(x, y);
        FMA_64(x, y);   FMA_64(x, y);   FMA_64(x, y);   FMA_64(x, y);
    }

    ptr[blockIdx.x * blockDim.x + threadIdx.x] = x + y;
}
