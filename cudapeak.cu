#include <iostream>
#include <hip/hip_runtime.h>
#include <cudaProfiler.h>

int main() {
    // Read device number from envirionment
    char *cstr_deviceNumber = getenv("CUDA_DEVICE");
    unsigned deviceNumber = cstr_deviceNumber ? atoi (cstr_deviceNumber) : 0;

    //  Setup CUDA
    hipSetDevice(deviceNumber);
    hipStream_t stream;
    hipStreamCreate(&stream);

    // Print CUDA device information
    hipDeviceProp_t deviceProperties;
    hipGetDeviceProperties(&deviceProperties, deviceNumber);
    std::cout << "Device " << deviceNumber << ": " << deviceProperties.name << std::endl;

    // Run benchmarks
    hipProfilerStart();
    hipProfilerStop();

    return EXIT_SUCCESS;
}
