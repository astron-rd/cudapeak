#include <iostream>
#include <string>
#include <iomanip>
#include <cstdio>
#include <cstdint>
#include <hip/hip_runtime.h>
#include <cudaProfiler.h>

#include "compute_sp_kernels.cu"
#include "mem_global_kernels.cu"

// Number of times to run each kernel
#define NR_ITERATIONS 10

// Number of times to run each benchmark
#define NR_BENCHMARKS 1

hipStream_t stream;
hipDeviceProp_t deviceProperties;

using namespace std;

void report(string name, double milliseconds, double gflops, double gbytes) {
    cout << setw(10) << string(name) << ": ";
    cout << setprecision(2) << fixed;
    cout << milliseconds << " ms ";
    if (gflops != 0)
        cout << ", " << gflops / milliseconds / 1e6 << " TFLOPS";
    if (gbytes != 0)
        cout << ", " << gbytes / milliseconds / 1e3 << " GB/s";
    cout << endl;
}

double run_kernel(
    void *kernel,
    float *ptr,
    dim3 gridDim,
    dim3 blockDim) {
    // Setup events
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Warmup
    ((void (*)(float *)) kernel)<<<gridDim, blockDim, 0, stream>>>(ptr);

    // Benchmark
    hipEventRecord(start, stream);
    for (int i = 0; i < NR_ITERATIONS; i++) {
        ((void (*)(float *)) kernel)<<<gridDim, blockDim, 0, stream>>>(ptr);
    }
    hipEventRecord(stop, stream);

    // Finish measurement
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    return milliseconds / NR_ITERATIONS;
}

void run_compute_sp() {
    // Parameters
    int multiProcessorCount = deviceProperties.multiProcessorCount;
    int maxThreadsPerBlock = deviceProperties.maxThreadsPerBlock;

    // Amount of work performed
    uint64_t nr_flops_block = 8192 * 2 * 1000;
    double nr_gflops_total = ((float) nr_flops_block * multiProcessorCount * maxThreadsPerBlock) / 1e3f;
    double nr_gybtes_total = 0;

    // Kernel dimensions
    dim3 gridDim(multiProcessorCount);
    dim3 blockDim(maxThreadsPerBlock);

    // Kernel data
    float *ptr;
    hipMalloc(&ptr, multiProcessorCount * maxThreadsPerBlock * sizeof(float));

    // Run kernel
    double milliseconds;
    milliseconds = run_kernel((void *) &compute_sp_v1, ptr, gridDim, blockDim);
    report("compute_sp_v1", milliseconds, nr_gflops_total, nr_gybtes_total);
}


void run_mem_global() {
    // Parameters
    int multiProcessorCount = deviceProperties.multiProcessorCount;
    int maxThreadsPerBlock = deviceProperties.maxThreadsPerBlock;

    // Amount of work performed
    uint64_t nr_bytes_block = 0;
    double nr_gbytes_total = ((float) nr_bytes_block * multiProcessorCount * maxThreadsPerBlock) / 1e3f;
    double nr_gflops_total = 0;

    // Kernel dimensions
    dim3 gridDim(multiProcessorCount);
    dim3 blockDim(maxThreadsPerBlock);

    // Kernel data
    float *ptr;
    hipMalloc(&ptr, multiProcessorCount * maxThreadsPerBlock * sizeof(float));

    // Run kernel
    double milliseconds;
    milliseconds = run_kernel((void *) &mem_global_v1, ptr, gridDim, blockDim);
    report("mem_global_v1", milliseconds, nr_gflops_total, nr_gbytes_total);
}


int main() {
    // Read device number from envirionment
    char *cstr_deviceNumber = getenv("CUDA_DEVICE");
    unsigned deviceNumber = cstr_deviceNumber ? atoi (cstr_deviceNumber) : 0;

    //  Setup CUDA
    hipSetDevice(deviceNumber);
    hipStreamCreate(&stream);
    hipGetDeviceProperties(&deviceProperties, deviceNumber);

    // Print CUDA device information
    std::cout << "Device " << deviceNumber << ": " << deviceProperties.name << std::endl;

    // Run benchmarks
    hipProfilerStart();
    for (int i = 0; i < NR_BENCHMARKS; i++) {
        run_compute_sp();
        run_mem_global();
    }
    hipProfilerStop();

    return EXIT_SUCCESS;
}
