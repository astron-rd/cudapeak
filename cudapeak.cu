#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include <iomanip>
#include <cstdio>
#include <cstdint>
#include <hip/hip_runtime.h>
#include <cudaProfiler.h>

#include "compute_sp_kernels.cu"
#include "compute_sp_ai_kernels.cu"
#include "mem_global_kernels.cu"


using namespace std;


// Number of times to run each kernel
#define NR_ITERATIONS 10

// Number of times to run each benchmark
#define NR_BENCHMARKS 1


// CUDA variables
hipStream_t stream;
hipDeviceProp_t deviceProperties;


void report(string name, double milliseconds, double gflops, double gbytes) {
    cout << setw(10) << string(name) << ": ";
    cout << setprecision(2) << fixed;
    cout << milliseconds << " ms";
    if (gflops != 0)
        cout << ", " << gflops / milliseconds / 1e3 << " TFLOPS";
    if (gbytes != 0)
        cout << ", " << gbytes / milliseconds << " GB/s";
    if (gflops != 0 && gbytes != 0) {
        float arithmetic_intensity = gflops / gbytes;
        cout << ", " << arithmetic_intensity << " Flop/byte";
    }
    cout << endl;
}


unsigned roundToPowOf2(unsigned number) {
    double logd = log(number) / log(2);
    logd = floor(logd);

    return (unsigned) pow(2, (int) logd);
}


double run_kernel(
    void *kernel,
    float *ptr,
    dim3 gridDim,
    dim3 blockDim) {
    // Setup events
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Warmup
    ((void (*)(float *)) kernel)<<<gridDim, blockDim, 0, stream>>>(ptr);

    // Benchmark
    hipEventRecord(start, stream);
    for (int i = 0; i < NR_ITERATIONS; i++) {
        ((void (*)(float *)) kernel)<<<gridDim, blockDim, 0, stream>>>(ptr);
    }
    hipEventRecord(stop, stream);

    // Finish measurement
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    return milliseconds / NR_ITERATIONS;
}


void run_compute_sp() {
    // Parameters
    int multiProcessorCount = deviceProperties.multiProcessorCount;
    int maxThreadsPerBlock = deviceProperties.maxThreadsPerBlock;

    // Amount of work performed
    double nr_gflops_total = 0;
    double nr_gbytes_total = 0;

    // Kernel dimensions
    dim3 gridDim(multiProcessorCount);
    dim3 blockDim(maxThreadsPerBlock);

    // Allocate memory
    float *ptr;
    hipMalloc(&ptr, multiProcessorCount * maxThreadsPerBlock * sizeof(float));

    // Run kernel
    double milliseconds;
    milliseconds = run_kernel((void *) &compute_sp_v1, ptr, gridDim, blockDim);
    nr_gflops_total = (1e-6 * multiProcessorCount * maxThreadsPerBlock) * (1ULL * 2 * 2 * 64 * 1024 * 8 * 8);
    nr_gbytes_total = 0;
    report("compute_sp_v1", milliseconds, nr_gflops_total, nr_gbytes_total);

    // Free memory
    hipFree(ptr);
}

void run_compute_sp_ai() {
    // Parameters
    int multiProcessorCount = deviceProperties.multiProcessorCount;
    int maxThreadsPerBlock = deviceProperties.maxThreadsPerBlock;

    // Amount of work performed
    double nr_gflops_total = (1e-6 * multiProcessorCount * maxThreadsPerBlock) * 1024;
    double nr_gbytes_total = (1e-6 * multiProcessorCount * maxThreadsPerBlock) * 1024 * 2 * sizeof(float);

    // Kernel dimensions
    dim3 gridDim(multiProcessorCount);
    dim3 blockDim(maxThreadsPerBlock);

    // Allocate memory
    float *ptr;
    hipMalloc(&ptr, multiProcessorCount * maxThreadsPerBlock * sizeof(float));

    // Run kernels
    double milliseconds;
    milliseconds = run_kernel((void *) &compute_sp_ai_v1, ptr, gridDim, blockDim);
    report("compute_sp_ai_v1", milliseconds, nr_gflops_total*2, nr_gbytes_total);

    milliseconds = run_kernel((void *) &compute_sp_ai_v2, ptr, gridDim, blockDim);
    report("compute_sp_ai_v2", milliseconds, nr_gflops_total*4, nr_gbytes_total);

    milliseconds = run_kernel((void *) &compute_sp_ai_v3, ptr, gridDim, blockDim);
    report("compute_sp_ai_v3", milliseconds, nr_gflops_total*6, nr_gbytes_total);

    milliseconds = run_kernel((void *) &compute_sp_ai_v4, ptr, gridDim, blockDim);
    report("compute_sp_ai_v4", milliseconds, nr_gflops_total*8, nr_gbytes_total);

    milliseconds = run_kernel((void *) &compute_sp_ai_v5, ptr, gridDim, blockDim);
    report("compute_sp_ai_v5", milliseconds, nr_gflops_total*16, nr_gbytes_total);

    milliseconds = run_kernel((void *) &compute_sp_ai_v6, ptr, gridDim, blockDim);
    report("compute_sp_ai_v6", milliseconds, nr_gflops_total*32, nr_gbytes_total);

    milliseconds = run_kernel((void *) &compute_sp_ai_v7, ptr, gridDim, blockDim);
    report("compute_sp_ai_v7", milliseconds, nr_gflops_total*40, nr_gbytes_total);

    // Free memory
    hipFree(ptr);
}


void run_mem_global() {
    // Parameters
    int maxThreadsPerBlock = deviceProperties.maxThreadsPerBlock;

    // Amount of work performed
    unsigned fetchPerBlock = 16;
    int maxItems = deviceProperties.totalGlobalMem / sizeof(float) / 2;
    int numItems = roundToPowOf2(maxItems);
    double nr_gbytes_total = (float) (numItems / fetchPerBlock) * sizeof(float) / 1e6;
    double nr_gflops_total = 0;

    // Kernel dimensions
    dim3 gridDim(numItems / (fetchPerBlock * maxThreadsPerBlock));
    dim3 blockDim(maxThreadsPerBlock);

    // Allocate memory
    float *ptr;
    hipMalloc(&ptr, numItems * sizeof(float));

    // Run kernel
    double milliseconds;
    milliseconds = run_kernel((void *) &mem_global_v1, ptr, gridDim, blockDim);
    report("mem_global_v1", milliseconds, nr_gflops_total, nr_gbytes_total);

    // Free memory
    hipFree(ptr);
}


int main() {
    // Read device number from envirionment
    char *cstr_deviceNumber = getenv("CUDA_DEVICE");
    unsigned deviceNumber = cstr_deviceNumber ? atoi (cstr_deviceNumber) : 0;

    //  Setup CUDA
    hipSetDevice(deviceNumber);
    hipStreamCreate(&stream);
    hipGetDeviceProperties(&deviceProperties, deviceNumber);

    // Print CUDA device information
    std::cout << "Device " << deviceNumber << ": " << deviceProperties.name << std::endl;

    // Run benchmarks
    hipProfilerStart();
    for (int i = 0; i < NR_BENCHMARKS; i++) {
        run_mem_global();
        run_compute_sp();
        run_compute_sp_ai();
    }
    hipProfilerStop();

    return EXIT_SUCCESS;
}
