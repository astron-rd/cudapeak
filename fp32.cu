#include "hip/hip_runtime.h"
#include "common.h"

__global__ void fp32_kernel(float *ptr);

void run(
    hipStream_t stream,
    hipDeviceProp_t deviceProperties)
{
    // Parameters
    int multiProcessorCount = deviceProperties.multiProcessorCount;
    int maxThreadsPerBlock = deviceProperties.maxThreadsPerBlock;

    // Amount of work performed
    int nr_iterations = 2048;
    double gflops = (1e-9 * multiProcessorCount * maxThreadsPerBlock) * (1ULL * nr_iterations * 8 * 4096);
    double gbytes = 0;

    // Kernel dimensions
    dim3 gridDim(multiProcessorCount);
    dim3 blockDim(maxThreadsPerBlock);

    // Allocate memory
    float *ptr;
    hipMalloc(&ptr, multiProcessorCount * maxThreadsPerBlock * sizeof(float));

    // Run kernel
    double milliseconds;
    milliseconds = run_kernel(stream, deviceProperties, (void *) &fp32_kernel, ptr, gridDim, blockDim);
    report("fp32", milliseconds, gflops, gbytes);

    // Free memory
    hipFree(ptr);
}
