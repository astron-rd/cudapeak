#include "hip/hip_runtime.h"

#define FMA_1(x, y)  asm("fma.rn.f32 %0, %1, %2, %3;" : "=f"(x) : "f"(x), "f"(y), "f"(x)); \
                     asm("fma.rn.f32 %0, %1, %2, %3;" : "=f"(y) : "f"(y), "f"(x), "f"(y));
#define FMA_4(x, y)  FMA_1(x, y)  FMA_1(x, y)  FMA_1(x, y)  FMA_1(x,y)
#define FMA_16(x, y) FMA_4(x, y)  FMA_4(x, y)  FMA_4(x, y)  FMA_4(x, y)
#define FMA_64(x, y) FMA_16(x, y) FMA_16(x, y) FMA_16(x, y) FMA_16(x, y)

#define SINCOS_1(x, y) asm("sin.approx.f32  %0, %1;" : "=f"(x) : "f"(x));\
                       asm("cos.approx.f32  %0, %1;" : "=f"(y) : "f"(y));
#define SINCOS_2(x, y)  SINCOS_1(x, y)  SINCOS_1(x, y)
#define SINCOS_4(x, y)  SINCOS_2(x, y)  SINCOS_2(x, y)
#define SINCOS_8(x, y)  SINCOS_4(x, y)  SINCOS_4(x, y)
#define SINCOS_16(x, y) SINCOS_8(x, y)  SINCOS_8(x, y)
#define SINCOS_32(x, y) SINCOS_16(x, y) SINCOS_16(x, y)
#define SINCOS_64(x, y) SINCOS_32(x, y) SINCOS_32(x, y)

__global__ void compute_sp_sincos_b0(float *ptr)
{
    float x = threadIdx.x;
    float y = 0;

    for (int i = 0; i < 1024; i++) {
        FMA_64(x, y);   FMA_64(x, y);   FMA_64(x, y);   FMA_64(x, y);
        FMA_64(x, y);   FMA_64(x, y);   FMA_64(x, y);   FMA_64(x, y);
        FMA_64(x, y);   FMA_64(x, y);   FMA_64(x, y);   FMA_64(x, y);
        FMA_64(x, y);   FMA_64(x, y);   FMA_64(x, y);   FMA_64(x, y);
        FMA_64(x, y);   FMA_64(x, y);   FMA_64(x, y);   FMA_64(x, y);
        FMA_64(x, y);   FMA_64(x, y);   FMA_64(x, y);   FMA_64(x, y);
        FMA_64(x, y);   FMA_64(x, y);   FMA_64(x, y);   FMA_64(x, y);
        FMA_64(x, y);   FMA_64(x, y);   FMA_64(x, y);   FMA_64(x, y);
    }

    ptr[blockIdx.x * blockDim.x + threadIdx.x] = x + y;
}

__global__ void compute_sp_sincos_b1(float *ptr)
{
    float x = threadIdx.x;
    float y = 0;

    for (int i = 0; i < 1024; i++) {
        SINCOS_64(x, y);   SINCOS_64(x, y);   SINCOS_64(x, y);   SINCOS_64(x, y);
        SINCOS_64(x, y);   SINCOS_64(x, y);   SINCOS_64(x, y);   SINCOS_64(x, y);
        SINCOS_64(x, y);   SINCOS_64(x, y);   SINCOS_64(x, y);   SINCOS_64(x, y);
        SINCOS_64(x, y);   SINCOS_64(x, y);   SINCOS_64(x, y);   SINCOS_64(x, y);
        SINCOS_64(x, y);   SINCOS_64(x, y);   SINCOS_64(x, y);   SINCOS_64(x, y);
        SINCOS_64(x, y);   SINCOS_64(x, y);   SINCOS_64(x, y);   SINCOS_64(x, y);
        SINCOS_64(x, y);   SINCOS_64(x, y);   SINCOS_64(x, y);   SINCOS_64(x, y);
        SINCOS_64(x, y);   SINCOS_64(x, y);   SINCOS_64(x, y);   SINCOS_64(x, y);
    }

    ptr[blockIdx.x * blockDim.x + threadIdx.x] = x + y;
}

__global__ void compute_sp_sincos_v1(float *ptr)
{
    float x = threadIdx.x;
    float y = 0;

    for (int i = 0; i < 1024; i++) {
        SINCOS_64(x, y); FMA_64(x, y); SINCOS_64(x, y); FMA_64(x, y);
        SINCOS_64(x, y); FMA_64(x, y); SINCOS_64(x, y); FMA_64(x, y);
        SINCOS_64(x, y); FMA_64(x, y); SINCOS_64(x, y); FMA_64(x, y);
        SINCOS_64(x, y); FMA_64(x, y); SINCOS_64(x, y); FMA_64(x, y);
        SINCOS_64(x, y); FMA_64(x, y); SINCOS_64(x, y); FMA_64(x, y);
        SINCOS_64(x, y); FMA_64(x, y); SINCOS_64(x, y); FMA_64(x, y);
        SINCOS_64(x, y); FMA_64(x, y); SINCOS_64(x, y); FMA_64(x, y);
        SINCOS_64(x, y); FMA_64(x, y); SINCOS_64(x, y); FMA_64(x, y);

        SINCOS_64(x, y); FMA_64(x, y); SINCOS_64(x, y); FMA_64(x, y);
        SINCOS_64(x, y); FMA_64(x, y); SINCOS_64(x, y); FMA_64(x, y);
        SINCOS_64(x, y); FMA_64(x, y); SINCOS_64(x, y); FMA_64(x, y);
        SINCOS_64(x, y); FMA_64(x, y); SINCOS_64(x, y); FMA_64(x, y);
        SINCOS_64(x, y); FMA_64(x, y); SINCOS_64(x, y); FMA_64(x, y);
        SINCOS_64(x, y); FMA_64(x, y); SINCOS_64(x, y); FMA_64(x, y);
        SINCOS_64(x, y); FMA_64(x, y); SINCOS_64(x, y); FMA_64(x, y);
        SINCOS_64(x, y); FMA_64(x, y); SINCOS_64(x, y); FMA_64(x, y);
    }

    ptr[blockIdx.x * blockDim.x + threadIdx.x] = y;
}

__global__ void compute_sp_sincos_v2(float *ptr)
{
    float x = threadIdx.x;
    float y = 0;

    for (int i = 0; i < 1024; i++) {
        SINCOS_32(x, y); FMA_64(x, y); SINCOS_32(x, y); FMA_64(x, y);
        SINCOS_32(x, y); FMA_64(x, y); SINCOS_32(x, y); FMA_64(x, y);
        SINCOS_32(x, y); FMA_64(x, y); SINCOS_32(x, y); FMA_64(x, y);
        SINCOS_32(x, y); FMA_64(x, y); SINCOS_32(x, y); FMA_64(x, y);
        SINCOS_32(x, y); FMA_64(x, y); SINCOS_32(x, y); FMA_64(x, y);
        SINCOS_32(x, y); FMA_64(x, y); SINCOS_32(x, y); FMA_64(x, y);
        SINCOS_32(x, y); FMA_64(x, y); SINCOS_32(x, y); FMA_64(x, y);
        SINCOS_32(x, y); FMA_64(x, y); SINCOS_32(x, y); FMA_64(x, y);

        SINCOS_32(x, y); FMA_64(x, y); SINCOS_32(x, y); FMA_64(x, y);
        SINCOS_32(x, y); FMA_64(x, y); SINCOS_32(x, y); FMA_64(x, y);
        SINCOS_32(x, y); FMA_64(x, y); SINCOS_32(x, y); FMA_64(x, y);
        SINCOS_32(x, y); FMA_64(x, y); SINCOS_32(x, y); FMA_64(x, y);
        SINCOS_32(x, y); FMA_64(x, y); SINCOS_32(x, y); FMA_64(x, y);
        SINCOS_32(x, y); FMA_64(x, y); SINCOS_32(x, y); FMA_64(x, y);
        SINCOS_32(x, y); FMA_64(x, y); SINCOS_32(x, y); FMA_64(x, y);
        SINCOS_32(x, y); FMA_64(x, y); SINCOS_32(x, y); FMA_64(x, y);
    }

    ptr[blockIdx.x * blockDim.x + threadIdx.x] = y;
}

__global__ void compute_sp_sincos_v3(float *ptr)
{
    float x = threadIdx.x;
    float y = 0;

    for (int i = 0; i < 1024; i++) {
        SINCOS_16(x, y); FMA_64(x, y); SINCOS_16(x, y); FMA_64(x, y);
        SINCOS_16(x, y); FMA_64(x, y); SINCOS_16(x, y); FMA_64(x, y);
        SINCOS_16(x, y); FMA_64(x, y); SINCOS_16(x, y); FMA_64(x, y);
        SINCOS_16(x, y); FMA_64(x, y); SINCOS_16(x, y); FMA_64(x, y);
        SINCOS_16(x, y); FMA_64(x, y); SINCOS_16(x, y); FMA_64(x, y);
        SINCOS_16(x, y); FMA_64(x, y); SINCOS_16(x, y); FMA_64(x, y);
        SINCOS_16(x, y); FMA_64(x, y); SINCOS_16(x, y); FMA_64(x, y);
        SINCOS_16(x, y); FMA_64(x, y); SINCOS_16(x, y); FMA_64(x, y);

        SINCOS_16(x, y); FMA_64(x, y); SINCOS_16(x, y); FMA_64(x, y);
        SINCOS_16(x, y); FMA_64(x, y); SINCOS_16(x, y); FMA_64(x, y);
        SINCOS_16(x, y); FMA_64(x, y); SINCOS_16(x, y); FMA_64(x, y);
        SINCOS_16(x, y); FMA_64(x, y); SINCOS_16(x, y); FMA_64(x, y);
        SINCOS_16(x, y); FMA_64(x, y); SINCOS_16(x, y); FMA_64(x, y);
        SINCOS_16(x, y); FMA_64(x, y); SINCOS_16(x, y); FMA_64(x, y);
        SINCOS_16(x, y); FMA_64(x, y); SINCOS_16(x, y); FMA_64(x, y);
        SINCOS_16(x, y); FMA_64(x, y); SINCOS_16(x, y); FMA_64(x, y);
    }

    ptr[blockIdx.x * blockDim.x + threadIdx.x] = y;
}

__global__ void compute_sp_sincos_v4(float *ptr)
{
    float x = threadIdx.x;
    float y = 0;

    for (int i = 0; i < 1024; i++) {
        SINCOS_8(x, y); FMA_64(x, y); SINCOS_8(x, y); FMA_64(x, y);
        SINCOS_8(x, y); FMA_64(x, y); SINCOS_8(x, y); FMA_64(x, y);
        SINCOS_8(x, y); FMA_64(x, y); SINCOS_8(x, y); FMA_64(x, y);
        SINCOS_8(x, y); FMA_64(x, y); SINCOS_8(x, y); FMA_64(x, y);
        SINCOS_8(x, y); FMA_64(x, y); SINCOS_8(x, y); FMA_64(x, y);
        SINCOS_8(x, y); FMA_64(x, y); SINCOS_8(x, y); FMA_64(x, y);
        SINCOS_8(x, y); FMA_64(x, y); SINCOS_8(x, y); FMA_64(x, y);
        SINCOS_8(x, y); FMA_64(x, y); SINCOS_8(x, y); FMA_64(x, y);

        SINCOS_8(x, y); FMA_64(x, y); SINCOS_8(x, y); FMA_64(x, y);
        SINCOS_8(x, y); FMA_64(x, y); SINCOS_8(x, y); FMA_64(x, y);
        SINCOS_8(x, y); FMA_64(x, y); SINCOS_8(x, y); FMA_64(x, y);
        SINCOS_8(x, y); FMA_64(x, y); SINCOS_8(x, y); FMA_64(x, y);
        SINCOS_8(x, y); FMA_64(x, y); SINCOS_8(x, y); FMA_64(x, y);
        SINCOS_8(x, y); FMA_64(x, y); SINCOS_8(x, y); FMA_64(x, y);
        SINCOS_8(x, y); FMA_64(x, y); SINCOS_8(x, y); FMA_64(x, y);
        SINCOS_8(x, y); FMA_64(x, y); SINCOS_8(x, y); FMA_64(x, y);
    }

    ptr[blockIdx.x * blockDim.x + threadIdx.x] = y;
}

__global__ void compute_sp_sincos_v5(float *ptr)
{
    float x = threadIdx.x;
    float y = 0;

    for (int i = 0; i < 1024; i++) {
        SINCOS_4(x, y); FMA_64(x, y); SINCOS_4(x, y); FMA_64(x, y);
        SINCOS_4(x, y); FMA_64(x, y); SINCOS_4(x, y); FMA_64(x, y);
        SINCOS_4(x, y); FMA_64(x, y); SINCOS_4(x, y); FMA_64(x, y);
        SINCOS_4(x, y); FMA_64(x, y); SINCOS_4(x, y); FMA_64(x, y);
        SINCOS_4(x, y); FMA_64(x, y); SINCOS_4(x, y); FMA_64(x, y);
        SINCOS_4(x, y); FMA_64(x, y); SINCOS_4(x, y); FMA_64(x, y);
        SINCOS_4(x, y); FMA_64(x, y); SINCOS_4(x, y); FMA_64(x, y);
        SINCOS_4(x, y); FMA_64(x, y); SINCOS_4(x, y); FMA_64(x, y);

        SINCOS_4(x, y); FMA_64(x, y); SINCOS_4(x, y); FMA_64(x, y);
        SINCOS_4(x, y); FMA_64(x, y); SINCOS_4(x, y); FMA_64(x, y);
        SINCOS_4(x, y); FMA_64(x, y); SINCOS_4(x, y); FMA_64(x, y);
        SINCOS_4(x, y); FMA_64(x, y); SINCOS_4(x, y); FMA_64(x, y);
        SINCOS_4(x, y); FMA_64(x, y); SINCOS_4(x, y); FMA_64(x, y);
        SINCOS_4(x, y); FMA_64(x, y); SINCOS_4(x, y); FMA_64(x, y);
        SINCOS_4(x, y); FMA_64(x, y); SINCOS_4(x, y); FMA_64(x, y);
        SINCOS_4(x, y); FMA_64(x, y); SINCOS_4(x, y); FMA_64(x, y);
    }

    ptr[blockIdx.x * blockDim.x + threadIdx.x] = y;
}

__global__ void compute_sp_sincos_v6(float *ptr)
{
    float x = threadIdx.x;
    float y = 0;

    for (int i = 0; i < 1024; i++) {
        SINCOS_2(x, y); FMA_64(x, y); SINCOS_2(x, y); FMA_64(x, y);
        SINCOS_2(x, y); FMA_64(x, y); SINCOS_2(x, y); FMA_64(x, y);
        SINCOS_2(x, y); FMA_64(x, y); SINCOS_2(x, y); FMA_64(x, y);
        SINCOS_2(x, y); FMA_64(x, y); SINCOS_2(x, y); FMA_64(x, y);
        SINCOS_2(x, y); FMA_64(x, y); SINCOS_2(x, y); FMA_64(x, y);
        SINCOS_2(x, y); FMA_64(x, y); SINCOS_2(x, y); FMA_64(x, y);
        SINCOS_2(x, y); FMA_64(x, y); SINCOS_2(x, y); FMA_64(x, y);
        SINCOS_2(x, y); FMA_64(x, y); SINCOS_2(x, y); FMA_64(x, y);

        SINCOS_2(x, y); FMA_64(x, y); SINCOS_2(x, y); FMA_64(x, y);
        SINCOS_2(x, y); FMA_64(x, y); SINCOS_2(x, y); FMA_64(x, y);
        SINCOS_2(x, y); FMA_64(x, y); SINCOS_2(x, y); FMA_64(x, y);
        SINCOS_2(x, y); FMA_64(x, y); SINCOS_2(x, y); FMA_64(x, y);
        SINCOS_2(x, y); FMA_64(x, y); SINCOS_2(x, y); FMA_64(x, y);
        SINCOS_2(x, y); FMA_64(x, y); SINCOS_2(x, y); FMA_64(x, y);
        SINCOS_2(x, y); FMA_64(x, y); SINCOS_2(x, y); FMA_64(x, y);
        SINCOS_2(x, y); FMA_64(x, y); SINCOS_2(x, y); FMA_64(x, y);
    }

    ptr[blockIdx.x * blockDim.x + threadIdx.x] = y;
}

__global__ void compute_sp_sincos_v7(float *ptr)
{
    float x = threadIdx.x;
    float y = 0;

    for (int i = 0; i < 1024; i++) {
        SINCOS_1(x, y); FMA_64(x, y); SINCOS_1(x, y); FMA_64(x, y);
        SINCOS_1(x, y); FMA_64(x, y); SINCOS_1(x, y); FMA_64(x, y);
        SINCOS_1(x, y); FMA_64(x, y); SINCOS_1(x, y); FMA_64(x, y);
        SINCOS_1(x, y); FMA_64(x, y); SINCOS_1(x, y); FMA_64(x, y);
        SINCOS_1(x, y); FMA_64(x, y); SINCOS_1(x, y); FMA_64(x, y);
        SINCOS_1(x, y); FMA_64(x, y); SINCOS_1(x, y); FMA_64(x, y);
        SINCOS_1(x, y); FMA_64(x, y); SINCOS_1(x, y); FMA_64(x, y);
        SINCOS_1(x, y); FMA_64(x, y); SINCOS_1(x, y); FMA_64(x, y);

        SINCOS_1(x, y); FMA_64(x, y); SINCOS_1(x, y); FMA_64(x, y);
        SINCOS_1(x, y); FMA_64(x, y); SINCOS_1(x, y); FMA_64(x, y);
        SINCOS_1(x, y); FMA_64(x, y); SINCOS_1(x, y); FMA_64(x, y);
        SINCOS_1(x, y); FMA_64(x, y); SINCOS_1(x, y); FMA_64(x, y);
        SINCOS_1(x, y); FMA_64(x, y); SINCOS_1(x, y); FMA_64(x, y);
        SINCOS_1(x, y); FMA_64(x, y); SINCOS_1(x, y); FMA_64(x, y);
        SINCOS_1(x, y); FMA_64(x, y); SINCOS_1(x, y); FMA_64(x, y);
        SINCOS_1(x, y); FMA_64(x, y); SINCOS_1(x, y); FMA_64(x, y);
    }

    ptr[blockIdx.x * blockDim.x + threadIdx.x] = y;
}


__global__ void compute_sp_sincos_v8(float *ptr)
{
    float x = threadIdx.x;
    float y = 0;

    for (int i = 0; i < 1024; i++) {
        SINCOS_1(x, y); FMA_64(x, y); FMA_64(x, y);
        SINCOS_1(x, y); FMA_64(x, y); FMA_64(x, y);
        SINCOS_1(x, y); FMA_64(x, y); FMA_64(x, y);
        SINCOS_1(x, y); FMA_64(x, y); FMA_64(x, y);
        SINCOS_1(x, y); FMA_64(x, y); FMA_64(x, y);
        SINCOS_1(x, y); FMA_64(x, y); FMA_64(x, y);
        SINCOS_1(x, y); FMA_64(x, y); FMA_64(x, y);
        SINCOS_1(x, y); FMA_64(x, y); FMA_64(x, y);

        SINCOS_1(x, y); FMA_64(x, y); FMA_64(x, y);
        SINCOS_1(x, y); FMA_64(x, y); FMA_64(x, y);
        SINCOS_1(x, y); FMA_64(x, y); FMA_64(x, y);
        SINCOS_1(x, y); FMA_64(x, y); FMA_64(x, y);
        SINCOS_1(x, y); FMA_64(x, y); FMA_64(x, y);
        SINCOS_1(x, y); FMA_64(x, y); FMA_64(x, y);
        SINCOS_1(x, y); FMA_64(x, y); FMA_64(x, y);
        SINCOS_1(x, y); FMA_64(x, y); FMA_64(x, y);
    }

    ptr[blockIdx.x * blockDim.x + threadIdx.x] = y;
}

__global__ void compute_sp_sincos_v9(float *ptr)
{
    float x = threadIdx.x;
    float y = 0;

    for (int i = 0; i < 1024; i++) {
        SINCOS_1(x, y); FMA_64(x, y); FMA_64(x, y); FMA_64(x, y); FMA_64(x, y);
        SINCOS_1(x, y); FMA_64(x, y); FMA_64(x, y); FMA_64(x, y); FMA_64(x, y);
        SINCOS_1(x, y); FMA_64(x, y); FMA_64(x, y); FMA_64(x, y); FMA_64(x, y);
        SINCOS_1(x, y); FMA_64(x, y); FMA_64(x, y); FMA_64(x, y); FMA_64(x, y);
        SINCOS_1(x, y); FMA_64(x, y); FMA_64(x, y); FMA_64(x, y); FMA_64(x, y);
        SINCOS_1(x, y); FMA_64(x, y); FMA_64(x, y); FMA_64(x, y); FMA_64(x, y);
        SINCOS_1(x, y); FMA_64(x, y); FMA_64(x, y); FMA_64(x, y); FMA_64(x, y);
        SINCOS_1(x, y); FMA_64(x, y); FMA_64(x, y); FMA_64(x, y); FMA_64(x, y);
    }

    ptr[blockIdx.x * blockDim.x + threadIdx.x] = y;
}
