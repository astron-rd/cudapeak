#include "hip/hip_runtime.h"
#define NR_ITERATIONS 512

#include "cosisin.cu"

template<int nr_fma, int nr_sincos>
__device__ void fp32_sincos_lu_1_1(float2& a, float2& b, float2& c)
{
    // Perform nr_fma * 2 fma
    #pragma unroll nr_fma
    for (int i = 0; i < nr_fma; i++) {
        asm("fma.rn.f32 %0, %1, %2, %3;" : "=f"(a.x) : "f"(b.x),  "f"(c.x), "f"(a.x));
    }
    // Perform nr_sincos * 1 sincos
    #pragma unroll nr_sincos
    for (int i = 0; i < nr_sincos; i++) {
        cosisin(a.x, &b.x, &b.y);
    }
}

template<int nr_fma, int nr_sincos>
__device__ void fp32_sincos_lu_2_1(float2& a, float2& b, float2& c)
{
    // Perofrm nr_fma * 2 fma
    #pragma unroll nr_fma
    for (int i = 0; i < nr_fma; i++) {
        asm("fma.rn.f32 %0, %1, %2, %3;" : "=f"(a.x) : "f"(b.x),  "f"(c.x), "f"(a.x));
        asm("fma.rn.f32 %0, %1, %2, %3;" : "=f"(a.x) : "f"(-b.y), "f"(c.y), "f"(a.x));
    }
    // Perform nr_sincos * 1 sincos
    #pragma unroll nr_sincos
    for (int i = 0; i < nr_sincos; i++) {
        cosisin(a.x, &b.x, &b.y);
    }
}

template<int nr_fma, int nr_sincos>
__device__ void fp32_sincos_lu_4_1(float2& a, float2& b, float2& c)
{
    // Perofrm nr_fma * 4 fma
    #pragma unroll nr_fma
    for (int i = 0; i < nr_fma; i++) {
        asm("fma.rn.f32 %0, %1, %2, %3;" : "=f"(a.x) : "f"(b.x),  "f"(c.x), "f"(a.x));
        asm("fma.rn.f32 %0, %1, %2, %3;" : "=f"(a.x) : "f"(-b.y), "f"(c.y), "f"(a.x));
        asm("fma.rn.f32 %0, %1, %2, %3;" : "=f"(a.y) : "f"(b.x),  "f"(c.y), "f"(a.y));
        asm("fma.rn.f32 %0, %1, %2, %3;" : "=f"(a.y) : "f"(b.y),  "f"(c.x), "f"(a.y));
    }
    // Perform nr_sincos * 1 sincos
    #pragma unroll nr_sincos
    for (int i = 0; i < nr_sincos; i++) {
        cosisin(a.x, &b.x, &b.y);
    }
}

__global__ void fp32_sincos_lu_1_8(float *ptr)
{
    float2 a = make_float2(threadIdx.x, threadIdx.x + 1);
    float2 b = make_float2(1, 2);
    float2 c = make_float2(3, 4);

    for (int i = 0; i < NR_ITERATIONS; i++) {
        for (int j = 0; j < 8192; j++) {
            fp32_sincos_lu_1_1<1, 8>(a, b, c);
        }
    }

    ptr[blockIdx.x * blockDim.x + threadIdx.x] = a.x + a.y + b.x + b.y + c.x + c.y;
}

__global__ void fp32_sincos_lu_1_4(float *ptr)
{
    float2 a = make_float2(threadIdx.x, threadIdx.x + 1);
    float2 b = make_float2(1, 2);
    float2 c = make_float2(3, 4);

    for (int i = 0; i < NR_ITERATIONS; i++) {
        for (int j = 0; j < 8192; j++) {
            fp32_sincos_lu_1_1<1, 4>(a, b, c);
        }
    }

    ptr[blockIdx.x * blockDim.x + threadIdx.x] = a.x + a.y + b.x + b.y + c.x + c.y;
}

__global__ void fp32_sincos_lu_1_2(float *ptr)
{
    float2 a = make_float2(threadIdx.x, threadIdx.x + 1);
    float2 b = make_float2(1, 2);
    float2 c = make_float2(3, 4);

    for (int i = 0; i < NR_ITERATIONS; i++) {
        for (int j = 0; j < 8192; j++) {
            fp32_sincos_lu_1_1<1, 2>(a, b, c);
        }
    }

    ptr[blockIdx.x * blockDim.x + threadIdx.x] = a.x + a.y + b.x + b.y + c.x + c.y;
}

__global__ void fp32_sincos_lu_1_1(float *ptr)
{
    float2 a = make_float2(threadIdx.x, threadIdx.x + 1);
    float2 b = make_float2(1, 2);
    float2 c = make_float2(3, 4);

    for (int i = 0; i < NR_ITERATIONS; i++) {
        for (int j = 0; j < 8192; j++) {
            fp32_sincos_lu_1_1<1, 1>(a, b, c);
        }
    }

    ptr[blockIdx.x * blockDim.x + threadIdx.x] = a.x + a.y + b.x + b.y + c.x + c.y;
}

__global__ void fp32_sincos_lu_2_1(float *ptr)
{
    float2 a = make_float2(threadIdx.x, threadIdx.x + 1);
    float2 b = make_float2(1, 2);
    float2 c = make_float2(3, 4);

    for (int i = 0; i < NR_ITERATIONS; i++) {
        for (int j = 0; j < 4096; j++) {
            fp32_sincos_lu_2_1<1, 1>(a, b, c);
        }
    }

    ptr[blockIdx.x * blockDim.x + threadIdx.x] = a.x + a.y + b.x + b.y + c.x + c.y;
}

__global__ void fp32_sincos_lu_4_1(float *ptr)
{
    float2 a = make_float2(threadIdx.x, threadIdx.x + 1);
    float2 b = make_float2(1, 2);
    float2 c = make_float2(3, 4);

    for (int i = 0; i < NR_ITERATIONS; i++) {
        for (int j = 0; j < 2048; j++) {
            fp32_sincos_lu_4_1<1, 1>(a, b, c);
        }
    }

    ptr[blockIdx.x * blockDim.x + threadIdx.x] = a.x + a.y + b.x + b.y + c.x + c.y;
}

__global__ void fp32_sincos_lu_8_1(float *ptr)
{
    float2 a = make_float2(threadIdx.x, threadIdx.x + 1);
    float2 b = make_float2(1, 2);
    float2 c = make_float2(3, 4);

    for (int i = 0; i < NR_ITERATIONS; i++) {
        for (int j = 0; j < 1024; j++) {
            fp32_sincos_lu_4_1<2, 1>(a, b, c);
        }
    }

    ptr[blockIdx.x * blockDim.x + threadIdx.x] = a.x + a.y + b.x + b.y + c.x + c.y;
}

__global__ void fp32_sincos_lu_16_1(float *ptr)
{
    float2 a = make_float2(threadIdx.x, threadIdx.x + 1);
    float2 b = make_float2(1, 2);
    float2 c = make_float2(3, 4);

    for (int i = 0; i < NR_ITERATIONS; i++) {
        for (int j = 0; j < 512; j++) {
            fp32_sincos_lu_4_1<4, 1>(a, b, c);
        }
    }

    ptr[blockIdx.x * blockDim.x + threadIdx.x] = a.x + a.y + b.x + b.y + c.x + c.y;
}

__global__ void fp32_sincos_lu_32_1(float *ptr)
{
    float2 a = make_float2(threadIdx.x, threadIdx.x + 1);
    float2 b = make_float2(1, 2);
    float2 c = make_float2(3, 4);

    for (int i = 0; i < NR_ITERATIONS; i++) {
        for (int j = 0; j < 256; j++) {
            fp32_sincos_lu_4_1<8, 1>(a, b, c);
        }
    }

    ptr[blockIdx.x * blockDim.x + threadIdx.x] = a.x + a.y + b.x + b.y + c.x + c.y;
}

__global__ void fp32_sincos_lu_64_1(float *ptr)
{
    float2 a = make_float2(threadIdx.x, threadIdx.x + 1);
    float2 b = make_float2(1, 2);
    float2 c = make_float2(3, 4);

    for (int i = 0; i < NR_ITERATIONS; i++) {
        for (int j = 0; j < 128; j++) {
            fp32_sincos_lu_4_1<16, 1>(a, b, c);
        }
    }

    ptr[blockIdx.x * blockDim.x + threadIdx.x] = a.x + a.y + b.x + b.y + c.x + c.y;
}

__global__ void fp32_sincos_lu_128_1(float *ptr)
{
    float2 a = make_float2(threadIdx.x, threadIdx.x + 1);
    float2 b = make_float2(1, 2);
    float2 c = make_float2(3, 4);

    for (int i = 0; i < NR_ITERATIONS; i++) {
        for (int j = 0; j < 64; j++) {
            fp32_sincos_lu_4_1<32, 1>(a, b, c);
        }
    }

    ptr[blockIdx.x * blockDim.x + threadIdx.x] = a.x + a.y + b.x + b.y + c.x + c.y;
}
