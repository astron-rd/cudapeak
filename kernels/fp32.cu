
#include <hip/hip_runtime.h>
template<int nr_fp32>
__device__ void fp32_8(
    float2& a, float2& b, float2& c)
{
    // Perform nr_fp32 * 4 fma
    #pragma unroll nr_fp32
    for (int i = 0; i < nr_fp32; i++) {
        asm("fma.rn.f32 %0, %1, %2, %3;" : "=f"(a.x) : "f"(b.x),  "f"(c.x), "f"(a.x));
        asm("fma.rn.f32 %0, %1, %2, %3;" : "=f"(a.x) : "f"(-b.y), "f"(c.y), "f"(a.x));
        asm("fma.rn.f32 %0, %1, %2, %3;" : "=f"(a.y) : "f"(b.x),  "f"(c.y), "f"(a.y));
        asm("fma.rn.f32 %0, %1, %2, %3;" : "=f"(a.y) : "f"(b.y),  "f"(c.x), "f"(a.y));
    }
}

__global__ void fp32_kernel(float *ptr)
{
    float2 a = make_float2(threadIdx.x, threadIdx.x + 1);
    float2 b = make_float2(1, 2);
    float2 c = make_float2(3, 4);

    for (int i = 0; i < 2048; i++) {
        fp32_8<4096>(a, b, c);
    }

    ptr[blockIdx.x * blockDim.x + threadIdx.x] = a.x + a.y;
}
