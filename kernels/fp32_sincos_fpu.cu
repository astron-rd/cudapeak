
#include <hip/hip_runtime.h>
#define NR_REPETITIONS 512
#define NR_ITERATIONS 8192

template <int nr_fma, int nr_sincos>
__device__ void fp32_sincos_fpu_1_1(float2& a, float2& b, float2& c) {
  for (int i = 0; i < nr_fma; i++) {
    asm("fma.rn.f32 %0, %1, %2, %3;"
        : "=f"(a.x)
        : "f"(b.x), "f"(c.x), "f"(a.x));
  }
  for (int i = 0; i < nr_sincos; i++) {
    b.x = sinf(a.x);
    b.y = cosf(a.x);
  }
}

__global__ void fp32_sincos_fpu_1_8(float* ptr) {
  float2 a = make_float2(threadIdx.x, threadIdx.x + 1);
  float2 b = make_float2(1, 2);
  float2 c = make_float2(3, 4);

  for (int i = 0; i < NR_REPETITIONS; i++) {
    for (int j = 0; j < NR_ITERATIONS; j++) {
      fp32_sincos_fpu_1_1<1, 8>(a, b, c);
    }
  }

  ptr[blockIdx.x * blockDim.x + threadIdx.x] =
      a.x + a.y + b.x + b.y + c.x + c.y;
}

__global__ void fp32_sincos_fpu_1_4(float* ptr) {
  float2 a = make_float2(threadIdx.x, threadIdx.x + 1);
  float2 b = make_float2(1, 2);
  float2 c = make_float2(3, 4);

  for (int i = 0; i < NR_REPETITIONS; i++) {
    for (int j = 0; j < NR_ITERATIONS; j++) {
      fp32_sincos_fpu_1_1<1, 4>(a, b, c);
    }
  }

  ptr[blockIdx.x * blockDim.x + threadIdx.x] =
      a.x + a.y + b.x + b.y + c.x + c.y;
}

__global__ void fp32_sincos_fpu_1_2(float* ptr) {
  float2 a = make_float2(threadIdx.x, threadIdx.x + 1);
  float2 b = make_float2(1, 2);
  float2 c = make_float2(3, 4);

  for (int i = 0; i < NR_REPETITIONS; i++) {
    for (int j = 0; j < NR_ITERATIONS; j++) {
      fp32_sincos_fpu_1_1<1, 2>(a, b, c);
    }
  }

  ptr[blockIdx.x * blockDim.x + threadIdx.x] =
      a.x + a.y + b.x + b.y + c.x + c.y;
}

__global__ void fp32_sincos_fpu_1_1(float* ptr) {
  float2 a = make_float2(threadIdx.x, threadIdx.x + 1);
  float2 b = make_float2(1, 2);
  float2 c = make_float2(3, 4);

  for (int i = 0; i < NR_REPETITIONS; i++) {
    for (int j = 0; j < NR_ITERATIONS; j++) {
      fp32_sincos_fpu_1_1<1, 1>(a, b, c);
    }
  }

  ptr[blockIdx.x * blockDim.x + threadIdx.x] =
      a.x + a.y + b.x + b.y + c.x + c.y;
}

__global__ void fp32_sincos_fpu_2_1(float* ptr) {
  float2 a = make_float2(threadIdx.x, threadIdx.x + 1);
  float2 b = make_float2(1, 2);
  float2 c = make_float2(3, 4);

  for (int i = 0; i < NR_REPETITIONS; i++) {
    for (int j = 0; j < NR_ITERATIONS / 2; j++) {
      fp32_sincos_fpu_1_1<2, 1>(a, b, c);
    }
  }

  ptr[blockIdx.x * blockDim.x + threadIdx.x] =
      a.x + a.y + b.x + b.y + c.x + c.y;
}

__global__ void fp32_sincos_fpu_4_1(float* ptr) {
  float2 a = make_float2(threadIdx.x, threadIdx.x + 1);
  float2 b = make_float2(1, 2);
  float2 c = make_float2(3, 4);

  for (int i = 0; i < NR_REPETITIONS; i++) {
    for (int j = 0; j < NR_ITERATIONS / 4; j++) {
      fp32_sincos_fpu_1_1<4, 1>(a, b, c);
    }
  }

  ptr[blockIdx.x * blockDim.x + threadIdx.x] =
      a.x + a.y + b.x + b.y + c.x + c.y;
}

__global__ void fp32_sincos_fpu_8_1(float* ptr) {
  float2 a = make_float2(threadIdx.x, threadIdx.x + 1);
  float2 b = make_float2(1, 2);
  float2 c = make_float2(3, 4);

  for (int i = 0; i < NR_REPETITIONS; i++) {
    for (int j = 0; j < NR_ITERATIONS / 8; j++) {
      fp32_sincos_fpu_1_1<8, 1>(a, b, c);
    }
  }

  ptr[blockIdx.x * blockDim.x + threadIdx.x] =
      a.x + a.y + b.x + b.y + c.x + c.y;
}

__global__ void fp32_sincos_fpu_16_1(float* ptr) {
  float2 a = make_float2(threadIdx.x, threadIdx.x + 1);
  float2 b = make_float2(1, 2);
  float2 c = make_float2(3, 4);

  for (int i = 0; i < NR_REPETITIONS; i++) {
    for (int j = 0; j < NR_ITERATIONS / 16; j++) {
      fp32_sincos_fpu_1_1<16, 1>(a, b, c);
    }
  }

  ptr[blockIdx.x * blockDim.x + threadIdx.x] =
      a.x + a.y + b.x + b.y + c.x + c.y;
}

__global__ void fp32_sincos_fpu_32_1(float* ptr) {
  float2 a = make_float2(threadIdx.x, threadIdx.x + 1);
  float2 b = make_float2(1, 2);
  float2 c = make_float2(3, 4);

  for (int i = 0; i < NR_REPETITIONS; i++) {
    for (int j = 0; j < NR_ITERATIONS / 32; j++) {
      fp32_sincos_fpu_1_1<32, 1>(a, b, c);
    }
  }

  ptr[blockIdx.x * blockDim.x + threadIdx.x] =
      a.x + a.y + b.x + b.y + c.x + c.y;
}

__global__ void fp32_sincos_fpu_64_1(float* ptr) {
  float2 a = make_float2(threadIdx.x, threadIdx.x + 1);
  float2 b = make_float2(1, 2);
  float2 c = make_float2(3, 4);

  for (int i = 0; i < NR_REPETITIONS; i++) {
    for (int j = 0; j < NR_ITERATIONS / 64; j++) {
      fp32_sincos_fpu_1_1<64, 1>(a, b, c);
    }
  }

  ptr[blockIdx.x * blockDim.x + threadIdx.x] =
      a.x + a.y + b.x + b.y + c.x + c.y;
}

__global__ void fp32_sincos_fpu_128_1(float* ptr) {
  float2 a = make_float2(threadIdx.x, threadIdx.x + 1);
  float2 b = make_float2(1, 2);
  float2 c = make_float2(3, 4);

  for (int i = 0; i < NR_REPETITIONS; i++) {
    for (int j = 0; j < NR_ITERATIONS / 128; j++) {
      fp32_sincos_fpu_1_1<128, 1>(a, b, c);
    }
  }

  ptr[blockIdx.x * blockDim.x + threadIdx.x] =
      a.x + a.y + b.x + b.y + c.x + c.y;
}
