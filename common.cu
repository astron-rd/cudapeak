#include "hip/hip_runtime.h"
#include <thread>
#include <cxxopts.hpp>

#include "common.h"

inline void __checkCudaCall(hipError_t err, const char* file, int line) {
  if (err != hipSuccess) {
    std::cerr << "CUDA error in " << file << " at line " << line << ": "
              << hipGetErrorString(err) << std::endl;
    exit(EXIT_FAILURE);
  }
}

#define checkCudaCall(err) __checkCudaCall(err, __FILE__, __LINE__)

void report(string name, measurement measurement, double gflops, double gbytes,
            double gops) {
  double milliseconds = measurement.runtime;
  double power = measurement.power;
  int w1 = 20;
  int w2 = 7;
  cout << setw(w1) << string(name) << ": ";
  cout << setprecision(2) << fixed;
  cout << setw(w2) << milliseconds << " ms";
  double seconds = milliseconds * 1e-3;
  if (gflops != 0) {
    cout << ", " << setw(w2) << gflops / seconds * 1e-3 << " TFlops/s";
  }
  if (power > 1) {
    cout << ", " << setw(w2) << power << " W";
    if (gflops != 0) {
      cout << ", " << setw(w2) << gflops / seconds / power << " GFlops/W";
    }
    if (gops != 0) {
      cout << ", " << setw(w2) << gops / seconds / power << " GOps/W";
    }
  }
  if (gbytes != 0) {
    cout << ", " << setw(w2) << gbytes / seconds << " GB/s";
  }
  if (gflops != 0 && gbytes != 0) {
    float arithmetic_intensity = gflops / gbytes;
    cout << ", " << setw(w2) << arithmetic_intensity << " Flop/byte";
  }
  if (gops != 0) {
    cout << ", " << setw(w2) << gops / seconds * 1e-3 << " TOps/s";
  }
  cout << endl;
}

unsigned roundToPowOf2(unsigned number) {
  double logd = log(number) / log(2);
  logd = floor(logd);

  return (unsigned)pow(2, (int)logd);
}

cxxopts::Options setupCommandLineParser(const char* argv[]) {
  cxxopts::Options options(argv[0], "Benchmark for BeamFormerKernel");

  const unsigned NR_BENCHMARKS = 1;
  const unsigned NR_ITERATIONS = 1;
#if defined(HAVE_PMT)
  const unsigned MEASURE_POWER = false;
  const unsigned BENCHMARK_DURATION = 4000;  // ms
#endif
  const unsigned DEVICE_ID = 0;

  options.add_options()(
      "nr_benchmarks", "Number of benchmarks",
      cxxopts::value<unsigned>()->default_value(std::to_string(NR_BENCHMARKS)))(
      "nr_iterations", "Number of kernel iteration per benchmark",
      cxxopts::value<unsigned>()->default_value(std::to_string(NR_ITERATIONS)))(
#if defined(HAVE_PMT)
      "measure_power", "Measure power",
      cxxopts::value<bool>()->default_value(std::to_string(MEASURE_POWER)))(
      "benchmark_duration", "Approximate number of ms to run the benchmark",
      cxxopts::value<unsigned>()->default_value(
          std::to_string(BENCHMARK_DURATION)))(
#endif
      "device_id", "Device ID",
      cxxopts::value<unsigned>()->default_value(std::to_string(DEVICE_ID)))(
      "h,help", "Print help");

  return options;
}

cxxopts::ParseResult getCommandLineOptions(int argc, const char* argv[]) {
  cxxopts::Options options = setupCommandLineParser(argv);

  try {
    cxxopts::ParseResult result = options.parse(argc, argv);

    if (result.count("help")) {
      std::cout << options.help() << std::endl;
      exit(EXIT_SUCCESS);
    }

    return result;

  } catch (const cxxopts::exceptions::exception& e) {
    std::cerr << "Error parsing command-line options: " << e.what()
              << std::endl;
    exit(EXIT_FAILURE);
  }
}

Benchmark::Benchmark(int argc, const char* argv[]) {
  // Parse command-line options
  cxxopts::ParseResult results = getCommandLineOptions(argc, argv);
  const unsigned device_number = results["device_id"].as<unsigned>();
  nr_benchmarks_ = results["nr_benchmarks"].as<unsigned>();
  nr_iterations_ = results["nr_iterations"].as<unsigned>();
#if defined(HAVE_PMT)
  measure_power_ = results["measure_power"].as<bool>();
  benchmark_duration_ = results["benchmark_duration"].as<unsigned>();
#endif

  // Setup CUDA
  checkCudaCall(hipSetDevice(device_number));
  checkCudaCall(hipStreamCreate(&stream_));
  checkCudaCall(hipGetDeviceProperties(&device_properties_, device_number));
  checkCudaCall(hipEventCreate(&event_start_));
  checkCudaCall(hipEventCreate(&event_end_));
  checkCudaCall(hipDeviceSynchronize());

  // Print CUDA device information
  std::cout << "Device " << device_number << ": " << device_properties_.name;
  std::cout << " (" << device_properties_.multiProcessorCount << "SMs, ";
  std::cout << device_properties_.clockRate * 1e-6 << " Ghz)" << std::endl;

#if defined(HAVE_PMT)
  pm_ = std::move(pmt::Create("nvidia"));
#endif
}

Benchmark::~Benchmark() {
  if (data_) {
    checkCudaCall(hipFree(data_));
  }
  checkCudaCall(hipStreamSynchronize(stream_));
  checkCudaCall(hipStreamDestroy(stream_));
  checkCudaCall(hipEventDestroy(event_start_));
  checkCudaCall(hipEventDestroy(event_end_));
}

void Benchmark::allocate(size_t bytes) {
  if (data_) {
    checkCudaCall(hipFree(data_));
  }
  checkCudaCall(hipMalloc(&data_, bytes));
  checkCudaCall(hipMemsetAsync(data_, 1, bytes, stream_));
  data_bytes_ = bytes;
}

void Benchmark::run(void* kernel, dim3 grid, dim3 block, const char* name,
                    double gflops, double gbytes, double gops) {
  measurement measurement = run_kernel(kernel, grid, block);
  report(name, measurement, gflops, gbytes, gops);
}

measurement Benchmark::run_kernel(void* kernel, dim3 grid, dim3 block) {
// Benchmark with power measurement
#if defined(HAVE_PMT)
  if (measurePower()) {
    float milliseconds = 0;
    unsigned nr_iterations = 0;

    std::thread thread([&] {
      checkCudaCall(hipEventRecord(event_start_, stream_));
      ((void (*)(void*))kernel)<<<grid, block, 0, stream_>>>(data_);
      checkCudaCall(hipEventRecord(event_end_, stream_));
      checkCudaCall(hipEventSynchronize(event_end_));
      checkCudaCall(
          hipEventElapsedTime(&milliseconds, event_start_, event_end_));
      nr_iterations = benchmarkDuration() / milliseconds;
      checkCudaCall(hipEventRecord(event_start_, stream_));
      for (int i = 0; i < nr_iterations; i++) {
        ((void (*)(void*))kernel)<<<grid, block, 0, stream_>>>(data_);
      }
      checkCudaCall(hipEventRecord(event_end_, stream_));
      checkCudaCall(hipEventSynchronize(event_end_));
      checkCudaCall(
          hipEventElapsedTime(&milliseconds, event_start_, event_end_));
    });
    std::this_thread::sleep_for(
        std::chrono::milliseconds(int(0.5 * benchmarkDuration())));
    pmt::State state_start = pm_->Read();
    std::this_thread::sleep_for(
        std::chrono::milliseconds(int(0.2 * benchmarkDuration())));
    pmt::State state_end = pm_->Read();
    if (thread.joinable()) {
      thread.join();
    }

    measurement measurement;
    measurement.runtime = milliseconds / nr_iterations;
    measurement.power = pmt::PMT::watts(state_start, state_end);

    return measurement;
  }
#endif

  // Benchmark (timing only)
  checkCudaCall(hipEventRecord(event_start_, stream_));
  for (int i = 0; i < nrIterations(); i++) {
    ((void (*)(void*))kernel)<<<grid, block, 0, stream_>>>(data_);
  }
  checkCudaCall(hipEventRecord(event_end_, stream_));
  hipEventSynchronize(event_end_);
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, event_start_, event_end_);
  measurement measurement;
  measurement.runtime = milliseconds / nrIterations();
  measurement.power = 0;
  return measurement;
}
