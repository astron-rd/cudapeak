#include "hip/hip_runtime.h"
#include "common.h"

__global__ void fp32_smem_01(float *ptr);
__global__ void fp32_smem_02(float *ptr);
__global__ void fp32_smem_04(float *ptr);
__global__ void fp32_smem_08(float *ptr);
__global__ void fp32_smem_16(float *ptr);
__global__ void fp32_smem_32(float *ptr);

void run(
    hipStream_t stream,
    hipDeviceProp_t deviceProperties)
{
    // Parameters
    int multiProcessorCount = deviceProperties.multiProcessorCount;
    int maxThreadsPerBlock = deviceProperties.maxThreadsPerBlock;
    int maxBlocksPerSM = deviceProperties.major >= 5 ? 32 : 16;

    // Amount of work performed
    unsigned workPerBlock = 128 * 512 * 2;
    unsigned globalBlocks = multiProcessorCount * maxBlocksPerSM * maxThreadsPerBlock;
    double gflops = (1e-9 * globalBlocks * workPerBlock);
    double gbytes = (1e-9 * globalBlocks * workPerBlock) * 2;

    // Kernel dimensions
    dim3 gridDim(multiProcessorCount, maxBlocksPerSM);
    dim3 blockDim(maxThreadsPerBlock);

    // Allocate memory
    float *ptr;
    hipMalloc(&ptr, multiProcessorCount * maxThreadsPerBlock * sizeof(float));

    // Run kernels
    double milliseconds;
    milliseconds = run_kernel(stream, deviceProperties, (void *) &fp32_smem_01, ptr, gridDim, blockDim);
    report("flop:byte ->  1:1", milliseconds, gflops*1, gbytes);

    milliseconds = run_kernel(stream, deviceProperties, (void *) &fp32_smem_02, ptr, gridDim, blockDim);
    report("flop:byte ->  2:1", milliseconds, gflops*2, gbytes);

    milliseconds = run_kernel(stream, deviceProperties, (void *) &fp32_smem_04, ptr, gridDim, blockDim);
    report("flop:byte ->  4:1", milliseconds, gflops*4, gbytes);

    milliseconds = run_kernel(stream, deviceProperties, (void *) &fp32_smem_08, ptr, gridDim, blockDim);
    report("flop:byte ->  8:1", milliseconds, gflops*8, gbytes);

    milliseconds = run_kernel(stream, deviceProperties, (void *) &fp32_smem_16, ptr, gridDim, blockDim);
    report("flop:byte -> 16:1", milliseconds, gflops*16, gbytes);

    milliseconds = run_kernel(stream, deviceProperties, (void *) &fp32_smem_32, ptr, gridDim, blockDim);
    report("flop:byte -> 32:1", milliseconds, gflops*32, gbytes);

    // Free memory
    hipFree(ptr);
}
