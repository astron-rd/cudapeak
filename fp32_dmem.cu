#include "hip/hip_runtime.h"
#include "common.h"

__global__ void fp32_dmem_01(float4* ptr);
__global__ void fp32_dmem_02(float4* ptr);
__global__ void fp32_dmem_04(float4* ptr);
__global__ void fp32_dmem_08(float4* ptr);
__global__ void fp32_dmem_16(float4* ptr);
__global__ void fp32_dmem_32(float4* ptr);
__global__ void fp32_dmem_64(float4* ptr);
__global__ void fp32_dmem_128(float4* ptr);
__global__ void fp32_dmem_256(float4* ptr);

int main(int argc, char* argv[]) {
  Benchmark benchmark;

  // Parameters
  int maxThreadsPerBlock = benchmark.maxThreadsPerBlock();

  // Amount of work performed
  unsigned fetchPerBlock = 16;
  int maxItems = benchmark.totalGlobalMem() / sizeof(float4) / 2;
  int numItems = roundToPowOf2(maxItems);
  double gbytes = (float)(numItems / fetchPerBlock * 2) * sizeof(float4) / 1e9;
  double gflops = gbytes * 4;

  // Kernel dimensions
  dim3 grid(numItems / (fetchPerBlock * maxThreadsPerBlock));
  dim3 block(maxThreadsPerBlock);

  // Allocate memory
  benchmark.allocate(numItems * sizeof(float4));

  // Run benchmark
  for (int i = 0; i < NR_BENCHMARKS; i++) {
    benchmark.run(reinterpret_cast<void*>(&fp32_dmem_01), grid, block,
                  "flop:byte ->   1:1", 1*gflops, gbytes);
    benchmark.run(reinterpret_cast<void*>(&fp32_dmem_02), grid, block,
                  "flop:byte ->   2:1", 2*gflops, gbytes);
    benchmark.run(reinterpret_cast<void*>(&fp32_dmem_04), grid, block,
                  "flop:byte ->   4:1", 4*gflops, gbytes);
    benchmark.run(reinterpret_cast<void*>(&fp32_dmem_08), grid, block,
                  "flop:byte ->   8:1", 8*gflops, gbytes);
    benchmark.run(reinterpret_cast<void*>(&fp32_dmem_16), grid, block,
                  "flop:byte ->  16:1", 16*gflops, gbytes);
    benchmark.run(reinterpret_cast<void*>(&fp32_dmem_32), grid, block,
                  "flop:byte ->  32:1", 32*gflops, gbytes);
    benchmark.run(reinterpret_cast<void*>(&fp32_dmem_64), grid, block,
                  "flop:byte ->  64:1", 64*gflops, gbytes);
    benchmark.run(reinterpret_cast<void*>(&fp32_dmem_128), grid, block,
                  "flop:byte -> 128:1", 128*gflops, gbytes);
    benchmark.run(reinterpret_cast<void*>(&fp32_dmem_256), grid, block,
                  "flop:byte -> 256:1", 256*gflops, gbytes);
  }

  return EXIT_SUCCESS;
}
